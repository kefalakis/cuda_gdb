#include "hip/hip_runtime.h"
#include "main.h"
#include <cstdio>

__global__ void kernel()
{
    const auto thread_id = threadIdx.x + blockDim.x * threadIdx.y;
    printf("just testing");
}

void kernelLaunch()
{
    kernel<<<1, 5>>>();
}